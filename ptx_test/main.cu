#include "hip/hip_runtime.h"



#include <cstdio>
#include <cstdint>
#include <cstdlib>

extern "C" __device__ void ptxTestFunc(int* dst, const int* src);

static __global__ void ptxTestKernel(int* dst, const int* src)
{
    ptxTestFunc(dst, src);
}

int main(void)
{
    hipDeviceProp_t props{ };
    auto cudaStatus = hipGetDeviceProperties(&props, 0);
    if (cudaStatus != hipSuccess)
    {
        printf("hipGetDeviceProperties failed: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }

    auto const maxBlocksPerSM = props.maxBlocksPerMultiProcessor;
    auto const maxThreadsPerBlock = props.maxThreadsPerBlock;
    auto const maxThreadsPerSM = props.maxThreadsPerMultiProcessor;
    auto const regsPerSM = props.regsPerMultiprocessor;
    auto const sharedMemSizePerBlock = props.sharedMemPerBlock;
    auto const constMemSize = props.totalConstMem;

    printf("Current GPU: %s\n", props.name);
    printf("max blocks per SM: %d\n", maxBlocksPerSM);
    printf("max threads per block: %d\n", maxThreadsPerBlock);
    printf("max threads per SM: %d\n", maxThreadsPerSM);
    printf("registers per SM: %d\n", regsPerSM);
    printf("shared memroy size per block: %zuKB\n", sharedMemSizePerBlock / 1024);
    printf("constant memory size on the device: %zuKB\n", constMemSize / 1024);

    puts("\n======== ptxTestKernel kernel info ========");

    hipFuncAttributes funcAttrs{ };
    cudaStatus = hipFuncGetAttributes(&funcAttrs, reinterpret_cast<const void*>(ptxTestKernel));
    if (cudaStatus != hipSuccess)
    {
        printf("hipFuncGetAttributes failed: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
    printf("max threads per block: %d\n", funcAttrs.maxThreadsPerBlock);
    printf("number of registers by each thread: %d\n", funcAttrs.numRegs);
    printf("local memory size by each thread: %zu bytes\n", funcAttrs.localSizeBytes);
    printf("shared memory size per block: %zu bytes\n", funcAttrs.sharedSizeBytes);
    printf("constant memory size: %zu bytes\n", funcAttrs.constSizeBytes);
    puts("");

    constexpr int elemCount = 4096;
    int* hostSrc = new int[elemCount];
    for (int i = 0; i < elemCount; i++)
        hostSrc[i] = i + 1;

    int* devDst = nullptr;
    int* devSrc = nullptr;

    constexpr auto bufferSize = elemCount * sizeof(*hostSrc);

    do
    {
        cudaStatus = hipMalloc(&devDst, bufferSize);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc devDst failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(devDst, hostSrc, bufferSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy to devDst failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMalloc(&devSrc, bufferSize);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc devSrc failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(devSrc, hostSrc, bufferSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy to devSrc failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        constexpr int threadsPerBlock = 256;
        constexpr auto nBlocks = elemCount / threadsPerBlock;

        ptxTestKernel <<< nBlocks, threadsPerBlock >>> (devDst, devSrc);

        cudaStatus = hipMemcpy(hostSrc, devDst, bufferSize, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
            printf("hipMemcpy to hostSrc failed: %s\n", hipGetErrorString(cudaStatus));

        // result verification
        bool success = true;
        for (int i = 0; i < elemCount; i++)
        {
            if (hostSrc[i] != (i + 1) * 2)
            {
                success = false;
                break;
            }
        }
        printf("Is equal? %s\n", success ? "YES" : "NO");

    } while (false);

    if (hostSrc != nullptr)
        delete[] hostSrc;

    if (devDst != nullptr)
        hipFree(devDst);
    if (devSrc != nullptr)
        hipFree(devSrc);

    return 0;
}

